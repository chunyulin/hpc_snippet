#include <cstdio>
#include <cstdlib>
#include <random>
#include <vector>
#include <hip/hip_runtime.h>
#include <omp.h>

using namespace std;

/*
 * Retrieve device IDs for all CUDA devices in the current system.
 */
int getAllGpus(int **gid)
{
    int i;
    int nGpus;

    hipGetDeviceCount(&nGpus);

    *gid = (int *)malloc(sizeof(int) * nGpus);

    for (i = 0; i < nGpus; i++)
    {
        (*gid)[i] = i;
    }

    return nGpus;
}

//-------------------------------------------------------------------------------------------------------
// Function    :  get_cpuid
// Description :  Get the CPU ID
//
// Note        :  Work on both macOS and Linux systems
//-------------------------------------------------------------------------------------------------------
int get_cpuid()
{

// See https://stackoverflow.com/questions/33745364/sched-getcpu-equivalent-for-os-x
   int CPU;

   CPU = sched_getcpu();

   return CPU;

} // FUNCTION : get_cpuid

int main(void)
{
    int N_GPU;
    long N = 1L<<31;
//    long N = 1L<<3;
    double t_start, t_stop;
    int *gid;

    N_GPU = getAllGpus(&gid);

    printf("N = %ld .\n", N);
    printf("Total %d GPU(s) are available.\n", N_GPU);

    vector<float> A(N);
    vector<float> B(N);
    vector<float> C(N);

    t_start = omp_get_wtime();
    // initialization
    #pragma omp parallel num_threads( 12*N_GPU )
    {
        long seed = 149874015+omp_get_thread_num();
        mt19937 generator( seed );
        normal_distribution<float> norm(0.0, 1.0);

        #pragma omp master
        printf("Total %2d threads are initialized.\n", omp_get_num_threads());
        #pragma omp barrier
        printf("My CPU ID is %2d for Thread %2d\n", get_cpuid(), omp_get_thread_num());

        #pragma omp for
        for (long i = 0; i < N; i++)
        {
            A[i] = norm(generator);
            B[i] = norm(generator);
        }
    }

    // vector add
    #pragma omp parallel for num_threads( 12*N_GPU )
    for (long i = 0; i < N; i++)
    {
        C[i] = A[i] + B[i];
    }
    t_stop = omp_get_wtime();

//    for (long i = 0; i < N; i++)
//        printf("C[%ld] = %+.8e\n", i, C[i]);

    printf("--------------------------------------------------------------\n");
    printf("Total execution time is %.4e s .\n", t_stop - t_start);

    return EXIT_SUCCESS;
}

